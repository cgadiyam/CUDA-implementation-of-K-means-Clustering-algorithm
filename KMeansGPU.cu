#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <float.h>   //FLT_MAX
#include "KMeans.h"

__constant__ Vector2 Clusters[3];

__global__ void KMeansKernel( Datapoint* data, long n, int k )
{
		//Assignment of each data point to a cluster
		int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
		if(threadID < n)
		{
			float Min_Dist = FLT_MAX;
			int nearest_cluster = 0;
			data[threadID].altered = false;
			for(int j=0;j<k;j++)
			{
				if(data[threadID].p.distSq(Clusters[j]) < Min_Dist)
				{
					Min_Dist = data[threadID].p.distSq(Clusters[j]);
					nearest_cluster = j;
				}
			}
			if(nearest_cluster != data[threadID].cluster)
			{
				data[threadID].cluster = nearest_cluster;
				data[threadID].altered = true;
			}
		}
}

bool KMeansGPU( Datapoint* data, long n, Vector2* clusters, int k )
{
	hipError_t status;
	bool exit = false;
	int count;
	Vector2 Center;
	int bytes1 = k * sizeof(Vector2);
	hipMalloc((void**) &Clusters, bytes1);
	hipMemcpyToSymbol(HIP_SYMBOL(Clusters), clusters, bytes1, 0, hipMemcpyHostToDevice);
	Datapoint* DataSet;
	int bytes2 = n * sizeof(Datapoint);
	hipMalloc((void**) &DataSet, bytes2);
	
    //iterates until no data point changes its cluster
	while(!exit)
	{
		count = 0;
		exit = true;

		hipMemcpy(DataSet, data, bytes2, hipMemcpyHostToDevice);
		dim3 dimBlock(768, 1); 
		dim3 dimGrid((int)ceil((float)n/768), 1);
		KMeansKernel<<<dimGrid, dimBlock>>>(DataSet, n, k);
		// Wait for completion
		hipDeviceSynchronize();
		// Check for errors
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed: " << hipGetErrorString(status) << std::endl;
			hipFree(DataSet);
			return false;
		}
		// Retrieve the result matrix
		hipMemcpy(data, DataSet, bytes2, hipMemcpyDeviceToHost);

		//calculation of new center for all 3 clusters
		for(int i=0;i<k;i++)
		{
			count = 0;
			Center.x = 0;
			Center.y = 0;
			for(int j=0;j<n;j++)
			{
				if(data[j].cluster == i)
				{
					Center.x += data[j].p.x;
					Center.y += data[j].p.y;
					count++;
				}
			}
			if(count >0)
			{
				clusters[i].x = (Center.x)/count;
				clusters[i].y = (Center.y)/count;
			}
		}
		hipMemcpyToSymbol(HIP_SYMBOL(Clusters), clusters, bytes1, 0, hipMemcpyHostToDevice);
		for(int i=0;i<n;i++)
		{
			if(data[i].altered == true)
			{
				data[i].altered = false;
				exit = false;
			}
		}
	}

	hipFree(DataSet);
	// Success
	return true;
}